#pragma once
#include <hip/hip_runtime.h>

#ifdef _CUDA

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(cudaError_t code, const char* file, int line, bool abort = true)
{
	if (code != cudaSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", cudaGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__
void optimizedProductCuda(float* m1, float* m2, float* res, int i) {
	float a = m1[i * 16 + 0] * m2[0 * 4 + 0] + m1[i * 16 + 1] * m2[1 * 4 + 0] + m1[i * 16 + 2] * m2[2 * 4 + 0] + m2[3 * 4 + 0];
	float b = m1[i * 16 + 0] * m2[0 * 4 + 1] + m1[i * 16 + 1] * m2[1 * 4 + 1] + m1[i * 16 + 2] * m2[2 * 4 + 1] + m2[3 * 4 + 1];
	float c = m1[i * 16 + 0] * m2[0 * 4 + 2] + m1[i * 16 + 1] * m2[1 * 4 + 2] + m1[i * 16 + 2] * m2[2 * 4 + 2] + m2[3 * 4 + 2];
	float d = m1[i * 16 + 0] * m2[0 * 4 + 3] + m1[i * 16 + 1] * m2[1 * 4 + 3] + m1[i * 16 + 2] * m2[2 * 4 + 3] + m2[3 * 4 + 3];
	res[i * 16 + 0] = a;
	res[i * 16 + 1] = b;
	res[i * 16 + 2] = c;
	res[i * 16 + 3] = d;
	
	for (int j = 4; j < 16; j++) {
		res[i * 16 + j] = 0;
	}
}

__device__
float optimizedLengthCuda(float* m, int i) {
	return sqrtf(m[i * 16 + 0] * m[i * 16 + 0] + m[i * 16 + 1] * m[i * 16 + 1] + m[i * 16 + 2] * m[i * 16 + 2]);
}

__global__
void bnw(int n, Uint32* data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	if (i < n) {
		Uint32 p = data[i];
		Uint8 a = (Uint8)(p >> 24);
		Uint8 r = (Uint8)(p >> 16);
		Uint8 g = (Uint8)(p >> 8);
		Uint8 b = (Uint8)(p);
		//const Uint8 grayscale = ((Uint8)(p >> 8) + (Uint8)(p >> 16) + (Uint8)(p >> 24)) / 3;

		Uint8 rred = (r * .393) + (g * .769) + (b * .189);
		Uint8 rgreen = (r * .349) + (g * .686) + (b * .168);
		Uint8 rblue = (r * .272) + (g * .534) + (b * .131);
		Uint8 ra = a;
		//p = (grayscale << 24) + (grayscale << 16) + (grayscale << 8) + (Uint8)p;
		p = (ra << 24) + (rblue << 16) + (rgreen << 8) + (Uint8)rred;
		data[i] = p;
		i += stride;
	}
}

__global__
void get2dCuda(int n, float* m, bool* clip, int centerx, int centery,
	float far, float* viewMatrix, float* projectionMatrix, float* res) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	while (i < n) {
		clip[i] = false;
		m[i * 16 + 0 * 4 + 3] = 1;
		optimizedProductCuda(m, viewMatrix, m, i);
		// camera space
		if (optimizedLengthCuda(m, i) > far) {
			clip[i] = true;
			res[i * 3 + 0] = (int)m[i * 16 + 0 * 4 + 0];
			res[i * 3 + 1] = (int)m[i * 16 + 0 * 4 + 1];
			res[i * 3 + 2] = 0;
			i += stride;
			continue;
		}
		optimizedProductCuda(m, projectionMatrix, m, i);
		//homogeneous clip space
		int x = (int)(m[i * 16 + 0 * 4 + 0] / m[i * 16 + 0 * 4 + 3]);
		int y = (int)(m[i * 16 + 0 * 4 + 1] / m[i * 16 + 0 * 4 + 3]);
		//NDC space[-1,1]
		x += centerx;
		y += centery;
		//raster space
		res[i * 3 + 0] = x;
		res[i * 3 + 1] = y;
		res[i * 3 + 2] = m[i * 16 + 0 * 4 + 3];
		i += stride;
	}
	__syncthreads();
}

__device__
int sign(int p1x, int p1y, int p2x, int p2y, int p3x, int p3y) {
	//return (p1.x - p3.x) * (p2.y - p3.y) - (p2.x - p3.x) * (p1.y - p3.y);
	return (p2x - p1x) * (p3y - p1y) - (p2y - p1y) * (p3x - p1x);
}

__device__
Uint8 clamp0_255(int n) {
	int tmp = 255;
	tmp -= n;
	tmp >>= 31;
	tmp |= n;
	n >>= 31;
	n = ~n;
	n &= tmp;
	return n;
}

// concat�ner une valeur de pixel et un flottant
__device__
Uint8 concatF(const Uint8& a, const Uint8& b) {
	return clamp0_255(a + b);
}

__device__ float fatomicMin(float* addr, float value)
{
	float old = *addr, assumed;
	if (old <= value) return old;
	while (old != assumed)
	{
		assumed = old;
		old = atomicCAS((unsigned int*)addr, __float_as_int(assumed), __float_as_int(value));
	}
	return old;
}

__global__
void rasterizeCuda(int n, int minx, int miny, int maxx, int maxy,
	int triAx, int triAy, int triBx, int triBy, int triCx, int triCy,
	int bmpAx, int bmpAy, int bmpBx, int bmpBy, int bmpCx, int bmpCy,
	Uint32* globalTexture, Uint32* srcpixels, float* zbuf, int srcw, int dstw, int srch, int dsth,
	float lightIntensity, Uint8 lightColorR, Uint8 lightColorG, Uint8 lightColorB,
	float depthA, float depthB, float depthC, float depthABC) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	float bymincy = triBy - triCy;
	float cxminbx = triCx - triBx;
	float axmincx = triAx - triCx;
	float aymincy = triAy - triCy;
	float cyminay = triCy - triAy;
	float divisor = bymincy * axmincx + cxminbx * aymincy;
	bymincy /= divisor;
	cxminbx /= divisor;
	cyminay /= divisor;
	axmincx /= divisor;

	while (i < n) {
		int x = minx + (i % (maxx - minx));
		int y = miny + (i / (maxx - minx));
		if (x > maxx || y > maxy)return;// continue;
		int bary0 = sign(triBx, triBy, triCx, triCy, minx - 1, y) + (x - minx + 1) * -(triCy - triBy) * 1;
		int bary1 = sign(triCx, triCy, triAx, triAy, minx - 1, y) + (x - minx + 1) * -(triAy - triCy) * 1;
		int bary2 = sign(triAx, triAy, triBx, triBy, minx - 1, y) + (x - minx + 1) * -(triBy - triAy) * 1;
		if (!(((bary0 < 0) || (bary1 < 0) || (bary2 < 0)) && ((bary0 > 0) || (bary1 > 0) || (bary2 > 0)))) {
			// pre calculate some other values
			const float pxmincx = x - triCx;
			// calculate barycentric coordinates
			const float baryA = bymincy * pxmincx + cxminbx * (y - triCy);
			const float baryB = cyminay * pxmincx + axmincx * (y - triCy);
			const float baryC = 1.0f - baryA - baryB;
			// add perspective correction
			const float pixdepth = depthABC / (baryA * depthB * depthC + baryB * depthA * depthC + baryC * depthA * depthB);
			const float u_ = (bmpAx / depthA) * baryA + (bmpBx / depthB) * baryB + (bmpCx / depthC) * baryC;
			const float v_ = (bmpAy / depthA) * baryA + (bmpBy / depthB) * baryB + (bmpCy / depthC) * baryC;
			// set position of source-pixel and new-pixel depth
			const int resx = u_ * pixdepth;
			const int resy = v_ * pixdepth;
			// define pos in bitmap
			const int it = x + y * dstw;
			// clipping check & pixel depth check
			fatomicMin(&zbuf[it], pixdepth);/////////////////////////////////////////////////////////////////////////////////////////
			if (zbuf[it] != pixdepth || resy < 0 || resy >= srch || resx < 0 || resx >= srcw) {
				i += stride;
				continue;
			}
			//zbuf[it] = pixdepth;///////////////////////////////////////////////////////////////////////////////////////////////////
			// src pixel position
			const size_t indexsrc = (size_t)resy * (size_t)srcw + (size_t)resx;
			// get texture pixel values (dst)
			const Uint8 A = srcpixels[indexsrc];
			const Uint8 R = srcpixels[indexsrc] >> 8;
			const Uint8 G = srcpixels[indexsrc] >> 16;
			const Uint8 B = srcpixels[indexsrc] >> 24;
			//globalTexture[it] = (B << 24) + (G << 16) + (R << 8) + A;
			//globalTexture[it] = ((B+light) << 24) + ((G + light) << 16) + ((R + light) << 8) + A;
			//__syncthreads();
			globalTexture[it] = (concatF(lightIntensity * lightColorB, B) << 24)
				+ (concatF(lightIntensity * lightColorG, G) << 16)
				+ (concatF(lightIntensity * lightColorR, R) << 8)
				+ (A);
			//globalTexture[it] = (Maths::clamp0_255(255-pixdepth*800) << 24) + (Maths::clamp0_255(255 - pixdepth * 800) << 16) + (Maths::clamp0_255(255 - pixdepth * 800) << 8) + (255);
		}
		i += stride;
	}
}

#endif